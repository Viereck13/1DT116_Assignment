#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define SIZE 1024
#define CELLSIZE 5
#define SCALED_SIZE SIZE*CELLSIZE
#define WEIGHTSUM 273


// Kernel 1: Fade heatmap – multiply all of previous pixels by 20%.
__global__ void fadeHeatmapKernel(int* d_heatmap, int totalPixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<totalPixels) {
        d_heatmap[idx] = (int)(d_heatmap[idx]*0.80f+0.5f); // 0.5f is for rounding. 8.1+0.5=8.6 -> 8, but 8.6+0.5=9
        // printf("d_heatmap[%d] = %d\n", idx, d_heatmap[idx]);
    }
}


// Kernel 2: Add color intensity to the desired position using atomicAdd
__global__ void addAgentHeatKernel(int* d_heatmap, int size,
                                   const int* d_agentDesiredX,
                                   const int* d_agentDesiredY,
                                   int numAgents) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < numAgents) {
        int x = d_agentDesiredX[idx];
        int y = d_agentDesiredY[idx];
        // printf("Agent %d at (%d, %d)\n", idx, x, y);
        if (x >= 0 && x < size && y >= 0 && y < size) {
            atomicAdd(&d_heatmap[y*size+x], 40);
            // printf("Heatmap[%d][%d] = %d\n", x, y, d_heatmap[y * size + x]);
        }
    }
}

// Kernel 3: Limit heatmap values to 255
__global__ void limitHeatmapValueKernel(int* d_heatmap, int totalPixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalPixels && d_heatmap[idx] > 255) {
        d_heatmap[idx]=255;
    }
}

// Kernel 4: Scale heatmap
__global__ void scaleHeatmapKernel(const int* d_heatmap, int* d_scaledHeatmap,
                                   int size, int cellSize) {
    int scaledSize = size*cellSize;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < scaledSize && y < scaledSize) {
        int origX = x/cellSize;
        int origY = y/cellSize;
        d_scaledHeatmap[y * scaledSize + x] = d_heatmap[origY * size + origX];
        // printf("Scaled Heatmap[%d][%d] = %d\n", y, x, d_scaledHeatmap[y * scaledSize + x]);
    }
}

// Kernel 5: Blur filter using shared memory and a 5x5 Gaussian kernel.
// Each thread computes one output pixel (except near the borders).
__global__ void blurFilterKernel(const int* d_scaledHeatmap, int* d_blurredHeatmap, int scaledSize) {
    // Allocate shared memory: tile dimensions plus a 2-pixel halo (aura) on each side.
    extern __shared__ int tile[]; // Know its size at runtime from <<<..., ..., sharedMemSize, ...>>>
    int tileWidth = blockDim.x + 4; // extra columns for halo

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int globalX = blockIdx.x * blockDim.x + tx;
    int globalY = blockIdx.y * blockDim.y + ty;
    int sharedX = tx+2;
    int sharedY = ty+2;

    // Load central data.
    if (globalX < scaledSize && globalY < scaledSize) {
        tile[sharedY*tileWidth + sharedX] = d_scaledHeatmap[globalY*scaledSize + globalX];
    } else {
        tile[sharedY*tileWidth + sharedX] = 0;
    }

    // Load halo for left and right edges.
    // Get two pixels to the left of the block.
    if (tx<2) {
        int gx = globalX-2;
        if (gx >= 0 && globalY < scaledSize)
            tile[sharedY*tileWidth+tx] = d_scaledHeatmap[globalY*scaledSize+gx];
        else
            tile[sharedY*tileWidth+tx] = 0;
    }
    // Get two pixels to the right of the block.
    if (tx >= blockDim.x-2) {
        int gx = globalX+2;
        if (gx < scaledSize && globalY < scaledSize)
            tile[sharedY * tileWidth + sharedX + 2] = d_scaledHeatmap[globalY * scaledSize + gx];
        else
            tile[sharedY * tileWidth + sharedX + 2] = 0;
    }

    // Load halo data for top and bottom edges.
    // Get two pixels above the block.
    if (ty<2) {
        int gy = globalY-2;
        if (gy >= 0 && globalX < scaledSize)
            tile[ty*tileWidth + sharedX] = d_scaledHeatmap[gy*scaledSize + globalX];
        else
            tile[ty*tileWidth + sharedX] = 0;
    }
    // Get two pixels below the block.
    if (ty >= blockDim.y-2) {
        int gy = globalY+2;
        if (gy < scaledSize && globalX < scaledSize)
            tile[(sharedY+2) * tileWidth + sharedX] = d_scaledHeatmap[gy*scaledSize + globalX];
        else
            tile[(sharedY+2) * tileWidth + sharedX] = 0;
    }

    // Load corner halo data.
    // Top-left corner.
    if (tx<2 && ty<2) {
        int gx = globalX-2;
        int gy = globalY-2;
        if (gx >= 0 && gy >= 0)
            tile[ty*tileWidth+tx] = d_scaledHeatmap[gy*scaledSize+gx];
        else
            tile[ty*tileWidth+tx] = 0;
    }
    // Top-right corner.
    if (tx >= blockDim.x-2 && ty < 2) {
        int gx = globalX+2;
        int gy = globalY-2;
        if (gx < scaledSize && gy >= 0)
            tile[ty*tileWidth + sharedX +2] = d_scaledHeatmap[gy*scaledSize + gx];
        else
            tile[ty*tileWidth + sharedX +2] = 0;
    }
    // Bottom-left corner.
    if (tx < 2 && ty >= blockDim.y-2) {
        int gx = globalX-2;
        int gy = globalY+2;
        if (gx >= 0 && gy < scaledSize)
            tile[(sharedY+2) * tileWidth +tx] = d_scaledHeatmap[gy * scaledSize +gx];
        else
            tile[(sharedY+2) * tileWidth +tx] = 0;
    }
    // Bottom-right corner.
    if (tx >= blockDim.x-2 && ty >= blockDim.y-2) {
        int gx = globalX+2;
        int gy = globalY+2;
        if (gx < scaledSize && gy < scaledSize)
            tile[(sharedY+2) * tileWidth + sharedX+2] = d_scaledHeatmap[gy*scaledSize + gx];
        else
            tile[(sharedY+2) * tileWidth + sharedX+2] = 0;
    }
    __syncthreads();

    // Only process if within valid bounds (No index out of bound).
    if (globalX >= 2 && globalX < scaledSize-2 && globalY >= 2 && globalY < scaledSize-2) {
        int weights[5][5] = {
            { 1, 4, 7, 4, 1 },
            { 4, 16, 26, 16, 4 },
            { 7, 26, 41, 26, 7 },
            { 4, 16, 26, 16, 4 },
            { 1, 4, 7, 4, 1 }
        };
        int sum = 0;
        for (int ky=-2; ky<=2; ky++) {
            for (int kx=-2; kx<=2; kx++) {
                sum += weights[ky+2][kx+2] * tile[(sharedY+ky)*tileWidth + (sharedX+kx)];
            }
        }
        int value = sum / WEIGHTSUM;
        d_blurredHeatmap[globalY*scaledSize+globalX] = 0x00FF0000 | (value << 24);
    }
}

void updateHeatmapCUDAAsync(int* h_heatmap, int* h_scaledHeatmap, int* h_blurredHeatmap,
                            const int* h_agentDesiredX, const int* h_agentDesiredY, int numAgents,
                            hipStream_t stream)
{
    int totalPixels = SIZE * SIZE;
    size_t heatmapSizeBytes = totalPixels * sizeof(int);
    size_t scaledTotalPixels = SCALED_SIZE * SCALED_SIZE;
    size_t scaledSizeBytes = scaledTotalPixels * sizeof(int);

    // Device pointers
    int *d_heatmap = nullptr;
    int *d_scaledHeatmap = nullptr;
    int *d_blurredHeatmap = nullptr;
    int *d_agentDesiredX = nullptr;
    int *d_agentDesiredY = nullptr;

    hipMalloc((void**)&d_heatmap, heatmapSizeBytes);
    hipMalloc((void**)&d_scaledHeatmap, scaledSizeBytes);
    hipMalloc((void**)&d_blurredHeatmap, scaledSizeBytes);
    hipMalloc((void**)&d_agentDesiredX, numAgents * sizeof(int));
    hipMalloc((void**)&d_agentDesiredY, numAgents * sizeof(int));

    hipEvent_t start, stop;
    hipEvent_t start_first, stop_first;
    hipEvent_t start_second, stop_second;
    hipEvent_t start_third, stop_third;

    hipStream_t stream1, stream2, stream3, stream0;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream0);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start_first);
    hipEventCreate(&stop_first);
    hipEventCreate(&start_second);
    hipEventCreate(&stop_second);
    hipEventCreate(&start_third);
    hipEventCreate(&stop_third);

    hipEventRecord(start, stream0);
    // Copy initial heatmap to device memory asynchronously.
    hipEventRecord(start_first, stream1);
    hipMemcpyAsync(d_heatmap, h_heatmap, heatmapSizeBytes, hipMemcpyHostToDevice, stream1);
    hipEventRecord(stop_first, stream1);

    hipEventRecord(start_second, stream2);
    hipMemcpyAsync(d_agentDesiredX, h_agentDesiredX, numAgents * sizeof(int), hipMemcpyHostToDevice, stream2);
    hipEventRecord(stop_second, stream2);

    hipEventRecord(start_third, stream3);
    hipMemcpyAsync(d_agentDesiredY, h_agentDesiredY, numAgents * sizeof(int), hipMemcpyHostToDevice, stream3);
    hipEventRecord(stop_third, stream3);

    hipEventRecord(stop, stream0);

    hipEventSynchronize(stop_first);
    hipEventSynchronize(stop_second);
    hipEventSynchronize(stop_third);
    hipEventSynchronize(stop);
    
    float timeHeatmap = 0, timeAgentX = 0, timeAgentY = 0;
    hipEventElapsedTime(&timeHeatmap, start_first, stop_first);
    hipEventElapsedTime(&timeAgentX, start_second, stop_second);
    hipEventElapsedTime(&timeAgentY, start_third, stop_third);

    // printf("Copy heatmap time: %f ms\n", timeHeatmap);
    // printf("Copy agentDesiredX time: %f ms\n", timeAgentX);
    // printf("Copy agentDesiredY time: %f ms\n", timeAgentY);

    // Measure overall elapsed time.
    float overallTime = 0;
    hipEventElapsedTime(&overallTime, start, stop);
    // printf("Overall async copy time (using separate streams): %f ms\n", overallTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(start_first);
    hipEventDestroy(stop_first);
    hipEventDestroy(start_second);
    hipEventDestroy(stop_second);
    hipEventDestroy(start_third);
    hipEventDestroy(stop_third);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream0);

    int threadsPerBlock = 256; // divisible by 32 (warp size)
    int blocksForFade = (totalPixels+threadsPerBlock-1) / threadsPerBlock;
    int blocksForAgents = (numAgents+threadsPerBlock-1) / threadsPerBlock;
    dim3 blockDim2D(16, 16); 
    dim3 gridDim2D((SCALED_SIZE+blockDim2D.x -1) / blockDim2D.x,
                   (SCALED_SIZE+blockDim2D.y -1) / blockDim2D.y);
    // determine the number of grids by SCALED_SIZE/blockDim2D.x and SCALED_SIZE/blockDim2D.y
    // (SCALED_SIZE + blockDim2D.x - 1) / blockDim2D.x to allow for partial blocks
    size_t sharedMemSize = (blockDim2D.x + 4) * (blockDim2D.y + 4) * sizeof(int); // +4 for halo, 2 on each side

    hipEvent_t startTotal, stopTotal;
    hipEvent_t startFade, stopFade;
    hipEvent_t startHeatmapAdd, stopHeatmapAdd;
    hipEvent_t startLimit, stopLimit;
    hipEvent_t startScale, stopScale;
    hipEvent_t startBlur, stopBlur;
    hipEventCreate(&startTotal);
    hipEventCreate(&stopTotal);
    hipEventCreate(&startFade);
    hipEventCreate(&stopFade);
    hipEventCreate(&startHeatmapAdd);
    hipEventCreate(&stopHeatmapAdd);
    hipEventCreate(&startLimit);
    hipEventCreate(&stopLimit);
    hipEventCreate(&startScale);
    hipEventCreate(&stopScale);
    hipEventCreate(&startBlur);
    hipEventCreate(&stopBlur);

    hipEventRecord(startTotal, stream);

    // Multiple kernels added to the stream will be executed in order of their addition.
    // No need of cudaDeviceSynchronize() after each kernel launch.

    // Launch Kernel 1
    hipEventRecord(startFade, stream);
    fadeHeatmapKernel<<<blocksForFade, threadsPerBlock, 0, stream>>>(d_heatmap, totalPixels);
    hipEventRecord(stopFade, stream);
    hipEventSynchronize(stopFade);
    float msFade = 0;
    hipEventElapsedTime(&msFade, startFade, stopFade);
    // printf("Async Fade kernel time: %f ms\n", msFade);

    // Launch Kernel 2
    hipEventRecord(startHeatmapAdd, stream);
    addAgentHeatKernel<<<blocksForAgents, threadsPerBlock, 0, stream>>>(d_heatmap, SIZE, d_agentDesiredX, d_agentDesiredY, numAgents);
    hipEventRecord(stopHeatmapAdd, stream);
    hipEventSynchronize(stopHeatmapAdd);
    float msHeatmapAdd = 0;
    hipEventElapsedTime(&msHeatmapAdd, startHeatmapAdd, stopHeatmapAdd);
    // printf("Async Heatmap Add kernel time: %f ms\n", msHeatmapAdd);

    // Launch Kernel 3
    hipEventRecord(startLimit, stream);
    limitHeatmapValueKernel<<<blocksForFade, threadsPerBlock, 0, stream>>>(d_heatmap, totalPixels);
    hipEventRecord(stopLimit, stream);
    hipEventSynchronize(stopLimit);
    float msLimit = 0;
    hipEventElapsedTime(&msLimit, startLimit, stopLimit);
    // printf("Async Limit kernel time: %f ms\n", msLimit);

    // Launch Kernel 4
    hipEventRecord(startScale, stream);
    scaleHeatmapKernel<<<gridDim2D, blockDim2D, 0, stream>>>(d_heatmap, d_scaledHeatmap, SIZE, CELLSIZE);
    // dim3 blockDim2D(16, 16) tells CUDA that each block should have 16 threads along the x-dimension, 
    // 16 threads along the y-dimension, and 1 thread along the z-dimension
    // resulting in a total of 256 threads per block. These values are used
    // inside the kernel to determine each thread’s unique indices via threadIdx.x, threadIdx.y, and threadIdx.z.
    hipEventRecord(stopScale, stream);
    hipEventSynchronize(stopScale);
    float msScale = 0;
    hipEventElapsedTime(&msScale, startScale, stopScale);
    // printf("Async Scale kernel time: %f ms\n", msScale);
    
    // Launch Kernel 5
    hipEventRecord(startBlur, stream);
    blurFilterKernel<<<gridDim2D, blockDim2D, sharedMemSize, stream>>>(d_scaledHeatmap, d_blurredHeatmap, SCALED_SIZE);
    hipEventRecord(stopBlur, stream);
    hipEventSynchronize(stopBlur);
    float msBlur = 0;
    hipEventElapsedTime(&msBlur, startBlur, stopBlur);
    // printf("Async Blur kernel time: %f ms\n", msBlur);

    hipEventRecord(stopTotal, stream);
    hipEventSynchronize(stopTotal);
    float msTotal = 0;
    hipEventElapsedTime(&msTotal, startTotal, stopTotal);
    // printf("Async Total time: %f ms\n", msTotal);

    // Copy the final heatmap and blurred heatmap back to host memory asynchronously.
    hipMemcpyAsync(h_blurredHeatmap, d_blurredHeatmap, scaledSizeBytes, hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(h_heatmap, d_heatmap, heatmapSizeBytes, hipMemcpyDeviceToHost, stream);

    // Free device memory.
    hipStreamSynchronize(stream);
    // printf("---------Async kernel execution complete-----------------\n");
    hipFree(d_heatmap);
    hipFree(d_scaledHeatmap);
    hipFree(d_blurredHeatmap);
    hipFree(d_agentDesiredX);
    hipFree(d_agentDesiredY);

    hipEventDestroy(startTotal);
    hipEventDestroy(stopTotal);
    hipEventDestroy(startFade);
    hipEventDestroy(stopFade);
    hipEventDestroy(startHeatmapAdd);
    hipEventDestroy(stopHeatmapAdd);
    hipEventDestroy(startLimit);
    hipEventDestroy(stopLimit);
    hipEventDestroy(startScale);
    hipEventDestroy(stopScale);
    hipEventDestroy(startBlur);
    hipEventDestroy(stopBlur);
}