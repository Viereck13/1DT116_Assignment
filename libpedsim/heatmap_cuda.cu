#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define SIZE 1024
#define CELLSIZE 5
#define SCALED_SIZE SIZE*CELLSIZE
#define WEIGHTSUM 273

__global__ void fadeHeatmapKernel(int* d_heatmap, int totalPixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<totalPixels) {
        d_heatmap[idx] = (int)(d_heatmap[idx]*0.80f+0.5f); // 0.5f is for rounding. 8.1+0.5=8.6 -> 8, but 8.6+0.5=9
        // printf("d_heatmap[%d] = %d\n", idx, d_heatmap[idx]);
    }
}

__global__ void addAgentHeatKernel(int* d_heatmap, int size,
                                   const int* d_agentDesiredX,
                                   const int* d_agentDesiredY,
                                   int numAgents) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < numAgents) {
        int x = d_agentDesiredX[idx];
        int y = d_agentDesiredY[idx];
        // printf("Agent %d at (%d, %d)\n", idx, x, y);
        if (x >= 0 && x < size && y >= 0 && y < size) {
            atomicAdd(&d_heatmap[y*size+x], 40);
            // printf("Heatmap[%d][%d] = %d\n", x, y, d_heatmap[y * size + x]);
        }
    }
}

__global__ void limitHeatmapValueKernel(int* d_heatmap, int totalPixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalPixels && d_heatmap[idx] > 255) {
        d_heatmap[idx]=255;
    }
}

__global__ void scaleHeatmapKernel(const int* d_heatmap, int* d_scaledHeatmap,
                                   int size, int cellSize) {
    int scaledSize = size*cellSize;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < scaledSize && y < scaledSize) {
        int origX = x/cellSize;
        int origY = y/cellSize;
        d_scaledHeatmap[y*scaledSize+x] = d_heatmap[origY*size+origX];
        // printf("Scaled Heatmap[%d][%d] = %d\n", y, x, d_scaledHeatmap[y * scaledSize + x]);
    }
}

// Each thread computes one output pixel (except near the borders).
__global__ void blurFilterKernel(const int* d_scaledHeatmap, int* d_blurredHeatmap, int scaledSize) {
    // Allocate shared memory: tileSharedMem dimensions plus a 2-pixel halo (aura) on each side.
    extern __shared__ int tileSharedMem[]; // Know its size at runtime from <<<..., ..., sharedMemSize, ...>>>
    int tileSharedMemWidth = blockDim.x + 4; // extra columns for halo

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int globalX = blockIdx.x * blockDim.x + tx;
    int globalY = blockIdx.y * blockDim.y + ty;
    int sharedX = tx+2;
    int sharedY = ty+2;

    // Load central data.
    if (globalX < scaledSize && globalY < scaledSize) {
        tileSharedMem[sharedY*tileSharedMemWidth + sharedX] = d_scaledHeatmap[globalY*scaledSize + globalX];
    } else {
        tileSharedMem[sharedY*tileSharedMemWidth + sharedX] = 0;
    }

    // Load halo for left and right edges.
    // Get two pixels to the left of the block.
    if (tx<2) {
        int gx = globalX-2;
        if (gx >= 0 && globalY < scaledSize)
            tileSharedMem[sharedY*tileSharedMemWidth+tx] = d_scaledHeatmap[globalY*scaledSize+gx];
        else
            tileSharedMem[sharedY*tileSharedMemWidth+tx] = 0;
    }
    // Get two pixels to the right of the block.
    if (tx >= blockDim.x-2) {
        int gx = globalX+2;
        if (gx < scaledSize && globalY < scaledSize)
            tileSharedMem[sharedY * tileSharedMemWidth + sharedX + 2] = d_scaledHeatmap[globalY * scaledSize + gx];
        else
            tileSharedMem[sharedY * tileSharedMemWidth + sharedX + 2] = 0;
    }

    // Load halo data for top and bottom edges.
    // Get two pixels above the block.
    if (ty<2) {
        int gy = globalY-2;
        if (gy >= 0 && globalX < scaledSize)
            tileSharedMem[ty*tileSharedMemWidth + sharedX] = d_scaledHeatmap[gy*scaledSize + globalX];
        else
            tileSharedMem[ty*tileSharedMemWidth + sharedX] = 0;
    }
    // Get two pixels below the block.
    if (ty >= blockDim.y-2) {
        int gy = globalY+2;
        if (gy < scaledSize && globalX < scaledSize)
            tileSharedMem[(sharedY+2) * tileSharedMemWidth + sharedX] = d_scaledHeatmap[gy*scaledSize + globalX];
        else
            tileSharedMem[(sharedY+2) * tileSharedMemWidth + sharedX] = 0;
    }

    // Load corner halo data.
    // Top-left corner.
    if (tx<2 && ty<2) {
        int gx = globalX-2;
        int gy = globalY-2;
        if (gx >= 0 && gy >= 0)
            tileSharedMem[ty*tileSharedMemWidth+tx] = d_scaledHeatmap[gy*scaledSize+gx];
        else
            tileSharedMem[ty*tileSharedMemWidth+tx] = 0;
    }
    // Top-right corner.
    if (tx >= blockDim.x-2 && ty < 2) {
        int gx = globalX+2;
        int gy = globalY-2;
        if (gx < scaledSize && gy >= 0)
            tileSharedMem[ty*tileSharedMemWidth + sharedX +2] = d_scaledHeatmap[gy*scaledSize + gx];
        else
            tileSharedMem[ty*tileSharedMemWidth + sharedX +2] = 0;
    }
    // Bottom-left corner.
    if (tx < 2 && ty >= blockDim.y-2) {
        int gx = globalX-2;
        int gy = globalY+2;
        if (gx >= 0 && gy < scaledSize)
            tileSharedMem[(sharedY+2) * tileSharedMemWidth +tx] = d_scaledHeatmap[gy * scaledSize +gx];
        else
            tileSharedMem[(sharedY+2) * tileSharedMemWidth +tx] = 0;
    }
    // Bottom-right corner.
    if (tx >= blockDim.x-2 && ty >= blockDim.y-2) {
        int gx = globalX+2;
        int gy = globalY+2;
        if (gx < scaledSize && gy < scaledSize)
            tileSharedMem[(sharedY+2) * tileSharedMemWidth + sharedX+2] = d_scaledHeatmap[gy*scaledSize + gx];
        else
            tileSharedMem[(sharedY+2) * tileSharedMemWidth + sharedX+2] = 0;
    }
    __syncthreads();

    // Only process if within valid bounds (No index out of bound).
    if (globalX >= 2 && globalX < scaledSize-2 && globalY >= 2 && globalY < scaledSize-2) {
        int weights[5][5] = {
            { 1, 4, 7, 4, 1 },
            { 4, 16, 26, 16, 4 },
            { 7, 26, 41, 26, 7 },
            { 4, 16, 26, 16, 4 },
            { 1, 4, 7, 4, 1 }
        };
        int sum = 0;
        for (int ky=-2; ky<=2; ky++) {
            for (int kx=-2; kx<=2; kx++) {
                sum += weights[ky+2][kx+2] * tileSharedMem[(sharedY+ky)*tileSharedMemWidth + (sharedX+kx)];
            }
        }
        int value = sum / WEIGHTSUM;
        d_blurredHeatmap[globalY*scaledSize+globalX] = 0x00FF0000 | (value << 24);
    }
}

void updateHeatmapCUDAAsync(int* h_heatmap, int* h_scaledHeatmap, int* h_blurredHeatmap,
    const int* h_agentDesiredX, const int* h_agentDesiredY, int numAgents,
    hipStream_t stream)
{
    int totalPixels = SIZE*SIZE;
    size_t heatmapSizeBytes = totalPixels * sizeof(int);
    size_t scaledTotalPixels = SCALED_SIZE*SCALED_SIZE;
    size_t scaledSizeBytes = scaledTotalPixels * sizeof(int);

    // Device pointers
    int *d_heatmap = nullptr;
    int *d_scaledHeatmap = nullptr;
    int *d_blurredHeatmap = nullptr;
    int *d_agentDesiredX = nullptr;
    int *d_agentDesiredY = nullptr;

    hipMalloc((void**)&d_heatmap, heatmapSizeBytes);
    hipMalloc((void**)&d_scaledHeatmap, scaledSizeBytes);
    hipMalloc((void**)&d_blurredHeatmap, scaledSizeBytes);
    hipMalloc((void**)&d_agentDesiredX, numAgents * sizeof(int));
    hipMalloc((void**)&d_agentDesiredY, numAgents * sizeof(int));


    // Copy initial heatmap to device memory asynchronously.
    hipMemcpyAsync(d_heatmap, h_heatmap, heatmapSizeBytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_agentDesiredX, h_agentDesiredX, numAgents * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_agentDesiredY, h_agentDesiredY, numAgents * sizeof(int), hipMemcpyHostToDevice, stream);
    // cudaMemcpy(d_heatmap, h_heatmap, heatmapSizeBytes, cudaMemcpyHostToDevice);
    // cudaMemcpy(d_agentDesiredX, h_agentDesiredX, numAgents * sizeof(int), cudaMemcpyHostToDevice);
    // cudaMemcpy(d_agentDesiredY, h_agentDesiredY, numAgents * sizeof(int), cudaMemcpyHostToDevice);

    int threadsPerBlock = 1024; // divisible by 32 (warp size)
    int blocksForFade = (totalPixels+threadsPerBlock-1) / threadsPerBlock;
    int blocksForAgents = (numAgents+threadsPerBlock-1) / threadsPerBlock;
    dim3 blockDim2D(32, 32); 
    dim3 gridDim2D((SCALED_SIZE+blockDim2D.x -1) / blockDim2D.x,(SCALED_SIZE+blockDim2D.y -1) / blockDim2D.y);
    // determine the number of grids by SCALED_SIZE/blockDim2D.x and SCALED_SIZE/blockDim2D.y
    // (SCALED_SIZE + blockDim2D.x - 1) / blockDim2D.x to allow for partial blocks
    size_t sharedMemSize = (blockDim2D.x + 4) * (blockDim2D.y + 4) * sizeof(int); // +4 for halo, 2 on each side

    // Multiple kernels added to the stream will be executed in order of their addition.
    // No need of cudaDeviceSynchronize() after each kernel launch.

    float elapsedTime = 0.0f;
    hipEvent_t start, stop;
    hipEvent_t startAll, stopAll;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startAll);
    hipEventCreate(&stopAll);

    hipEventRecord(startAll, stream);

    hipEventRecord(start, stream);
    fadeHeatmapKernel<<<blocksForFade, threadsPerBlock, 0, stream>>>(d_heatmap, totalPixels);
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Fade Heatmap Kernel Execution Time: %f ms\n", elapsedTime);

    hipEventRecord(start, stream);
    addAgentHeatKernel<<<blocksForAgents, threadsPerBlock, 0, stream>>>(d_heatmap, SIZE, d_agentDesiredX, d_agentDesiredY, numAgents);
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Add Agent Heat Kernel Execution Time: %f ms\n", elapsedTime);

    hipEventRecord(start, stream);
    limitHeatmapValueKernel<<<blocksForFade, threadsPerBlock, 0, stream>>>(d_heatmap, totalPixels);
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Limit Heatmap Value Kernel Execution Time: %f ms\n", elapsedTime);

    hipEventRecord(start, stream);
    scaleHeatmapKernel<<<gridDim2D, blockDim2D, 0, stream>>>(d_heatmap, d_scaledHeatmap, SIZE, CELLSIZE);
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Scale Heatmap Kernel Execution Time: %f ms\n", elapsedTime);
    // dim3 blockDim2D(16, 16) tells CUDA that each block should have 16 threads along the x-dimension, 
    // 16 threads along the y-dimension, and 1 thread along the z-dimension
    // resulting in a total of 256 threads per block. These values are used
    // inside the kernel to determine each thread’s unique indices via threadIdx.x, threadIdx.y, and threadIdx.z.
    hipEventRecord(start, stream);
    blurFilterKernel<<<gridDim2D, blockDim2D, sharedMemSize, stream>>>(d_scaledHeatmap, d_blurredHeatmap, SCALED_SIZE);
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Blur Filter Kernel Execution Time: %f ms\n", elapsedTime);

    hipEventRecord(stopAll, stream);
    hipEventSynchronize(stopAll);
    hipEventElapsedTime(&elapsedTime, startAll, stopAll);
    printf("Total Async Kernel Execution Time: %f ms\n", elapsedTime);

    // // Launch Artificial Workload Kernel
    // int numElements = 1024;  // You can adjust this size if needed.
    // int *d_dummyData = nullptr;
    // cudaMalloc(&d_dummyData, numElements * sizeof(int));
    // int iterations = 1000000; // Adjust this number to increase the workload.
    // int blocks = (SIZE + threadsPerBlock - 1) / threadsPerBlock;
    // artificialWorkloadKernel<<<blocks, threadsPerBlock, 0, stream>>>(d_dummyData, iterations);

    // Copy the final heatmap and blurred heatmap back to host memory asynchronously.
    hipMemcpyAsync(h_blurredHeatmap, d_blurredHeatmap, scaledSizeBytes, hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(h_heatmap, d_heatmap, heatmapSizeBytes, hipMemcpyDeviceToHost, stream);
    // cudaMemcpy(h_blurredHeatmap, d_blurredHeatmap, scaledSizeBytes, cudaMemcpyDeviceToHost);
    // cudaMemcpy(h_heatmap, d_heatmap, heatmapSizeBytes, cudaMemcpyDeviceToHost);

    // Free device memory.
    // cudaStreamSynchronize(stream); // CPU waits for GPU to finish before CPU moves on to the next step.
    // printf("---------Async kernel execution complete-----------------\n");
    hipFree(d_heatmap);
    hipFree(d_scaledHeatmap);
    hipFree(d_blurredHeatmap);
    hipFree(d_agentDesiredX);
    hipFree(d_agentDesiredY);
}